#include "hip/hip_runtime.h"
//TO-DO
//1*  Finish checking the boundaries of the blocks to ignore computations out of bounds. DONE(for the leftmsot and top)
//1** Missing bottom and right vectors.
//Global reverse ty && tx


#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE_PER_DIM 16		/*Maybe just TILE_WIDTH 16 and use just that? Though I do want a 3v3 at a time.*/
#define NUM_OF_BLOCKS 16
#define HEIGHT 32
#define WIDTH  32

__device__ __constant__ double c11 = +0.2, c21 = +0.5, c31 = -0.8, c12 = -0.3, c22 = +0.6, c32 = -0.9, c13 = +0.4, c23 = +0.7, c33 = +0.10;

__global__ void convolution_with_cuda(double* A, double* B , int num_of_Blocks) {
	
	int i, j;							

	__shared__ double A_per_blk[BLOCK_SIZE_PER_DIM][BLOCK_SIZE_PER_DIM];		/*BLOCK_SIZE_PER_DIM OR TILE WIDHT?	Edw evala 17x17 everything I need for a 16x16*/
	__shared__ double 
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int Row = by * BLOCK_SIZE_PER_DIM + ty;
	int Col = bx * BLOCK_SIZE_PER_DIM + tx;

		A_per_blk[ty][tx] = A[Row * WIDTH + tx];
		__syncthreads();

		if ((Row < HEIGHT -1) && (Col < WIDTH -1) && (Row >= 1) && (Col >= 1) ){  // 1*
			if(tx - 1 == -1 && ty - 1 == -1){
				B = A[tx - 1][Row * WIDTH + tx]*c11	
			}	
			else{
				
			}
			B = A_per_blk[tx - 1][ty - 1]*c11
			+A_per_blk[tx][ty - 1]*c12
			+A_per_blk[tx + 1][ty - 1]*c13
			+A_per_blk[tx + 1][ty]*c23
			+A_per_blk[tx + 1][ty + 1]*c33
			+A_per_blk[tx][ty + 1]*c32
			+A_per_blk[tx - 1][ty + 1]*c31
			+A_per_blk[tx - 1][ty]*c21
			+A_per_blk[tx][ty]*c22;

			//Code to be executed
		}


	/*Want to iterate the blocks until I am done with the 3x3 matrices computations.*/

		/*	for (i = 1; i < tx_max - 1; ++i) {
				for (j = 1; j < NJ - 1; ++j) {
					B[i*NJ + j] = c11 * A[(i - 1)*NJ + (j - 1)] + c12 * A[(i + 0)*NJ + (j - 1)] + c13 * A[(i + 1)*NJ + (j - 1)]
						+ c21 * A[(i - 1)*NJ + (j + 0)] + c22 * A[(i + 0)*NJ + (j + 0)] + c23 * A[(i + 1)*NJ + (j + 0)]
						+ c31 * A[(i - 1)*NJ + (j + 1)] + c32 * A[(i + 0)*NJ + (j + 1)] + c33 * A[(i + 1)*NJ + (j + 1)];
				}
			}
		*/
	//}
}

void init(double* A)
{
	int i, j;

	for (i = 0; i < HEIGHT; ++i) {
		for (j = 0; j < WIDTH; ++j) {
			A[i*WIDTH + j] = (double)rand() / RAND_MAX;
		}
	}
}

int main(int argc, char* argv[]) {

	int i, j;
	size_t size = WIDTH * HEIGHT * sizeof(double);

	double* A;
	double* B;

	A = (double*)malloc(size);
	if (A == NULL){
		fprintf(stderr,"malloc() Failed");
		return -1;
	}

	B = (double*)calloc(WIDTH * HEIGHT,sizeof(double));
	if (B == NULL) {
		fprintf(stderr,"calloc() Failed");
		return -1;
	}
	
	//initialize the arrays
	init(A);

	for (i = 0; i < HEIGHT; ++i) {
		for (j = 0; j < WIDTH; ++j) {
			fprintf(stdout,"A[%d,%d] = %f\n", i, j, A[i*WIDTH + j]);
		}
	}
	/************************************************Preparing the kernel *****************************************************/
	/* First the dimensions of the grid and blocks , we want them however many blocks are divided but each with 16x16 threads!
	Due to the inversed notation in the Cuda Standard ( Linear : grammes x sthles , Cuda : sthles x grammes )*/
	
	double* d_A, *d_B;

	unsigned int n_Blocks_X = ((WIDTH - 1) / BLOCK_SIZE_PER_DIM ) + 1;
	unsigned int n_Blocks_Y = ((HEIGHT - 1) / BLOCK_SIZE_PER_DIM) + 1;

	int nBlocks = n_Blocks_X * n_Blocks_Y;

	dim3 threads_per_block(BLOCK_SIZE_PER_DIM, BLOCK_SIZE_PER_DIM, 1);

	dim3 num_of_Blocks(n_Blocks_X, n_Blocks_Y, 1);

	size_t bytes_per_block = 16 * 16 * 64 / 8;						/*Because we will need a 17x17 matrix of information * 64bit each (cause its a double) / by 8 (bytes)*/

	hipError_t cuda_status = hipMalloc((void **)&d_A, size);
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "hipMalloc Failed!");
		goto Error;
		}

	cuda_status = hipMalloc((void **)&d_B, size);
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "hipMalloc Failed!");
		goto Error;
	}

	// Copying the data to the device 
	cuda_status = hipMemcpy(d_A , A, size, hipMemcpyHostToDevice);
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "hipMemcpy Failed!");
		goto Error;
	}

	cuda_status = hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "hipMemcpy Failed!");
		goto Error;
	}

	// Choose which GPU to run on, change this on a multi-GPU system.
	cuda_status = hipSetDevice(0);
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}
	//Executing the kernel call 

	convolution_with_cuda<<< num_of_Blocks, threads_per_block>>>(d_A , d_B ,nBlocks);

	// Check for any errors launching the kernel
	cuda_status = hipGetLastError();
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "convolution_with_cuda launch failed: %s\n", hipGetErrorString(cuda_status));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cuda_status = hipDeviceSynchronize();
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching convolution_with_cuda!\n", cuda_status);

		goto Error;
	}

	//Copying the data back and freeing the allocated space.

	cuda_status = hipMemcpy(B, d_B, size, hipMemcpyDeviceToHost);
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "hipMemcpy Failed!");
		goto Error;
	}

	//Output the new array 
	for (i = 0; i < HEIGHT; ++i) {
		for (j = 0; j < WIDTH; ++j) {
			fprintf(stdout,"B[%d,%d] = %f\n", i, j, B[i*WIDTH + j]);
		}
	}

	Error :
	hipFree(d_A);
	hipFree(d_B);
	
	return 0;
}