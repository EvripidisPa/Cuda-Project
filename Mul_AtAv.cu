﻿/*TO DO 
	No major issues , the code could be optimised a little bit more. Specifically in the init of the array section.
	1** Remove the debugging printf commands.
	
	*/

#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include "hipblas.h"

#define IDX2C(i,j,ld) (((j)*(ld))+(i))		//CuBlas column-major storage , the array index of a matrix element in row “i” and column “j” can be computed via this macro 

/*Problem size*/

#define HEIGHT 4                                 //  number  of rows of a
#define WIDTH  3								 //  number  of  columns  of a

#ifndef M_PI
#define M_PI 3.14159
#endif

void init_array(double *x, double *y, double *A)
{
	int i, j;

	for (i = 0; i < WIDTH; i++) x[i] = i * M_PI;						// x={i * M_PI;}^T
	for (i = 0; i < HEIGHT; i++) y[i] = 0.0f;							// y={0...0}^T


	for (i = 0; i < HEIGHT; i++) {										
		for (j = 0; j < WIDTH; j++) {
			A[IDX2C(i, j, HEIGHT)] = ((double)i*(j)) / HEIGHT;
		}
	}
}

int main()
{
	hipError_t  cuda_status;                              //  hipMalloc  status
	hipblasStatus_t  cuBl_stat;                               //  CUBLAS  functions  status
	hipblasHandle_t  handle;                             //  CUBLAS  context

	size_t sizeA = WIDTH * HEIGHT * sizeof(double);
	size_t size_x = WIDTH * sizeof(double);
	size_t size_y = HEIGHT * sizeof(double);

	int i, j;

	double		*A;										//	A - HEIGHT x WIDTH matrix on the host
	double		*x;										//	x -  WIDTH vector the host
	double		*y; 									//	y -  HEIGHT vector the host

	A = (double*)malloc(sizeA);
	if (A == NULL) {
		fprintf(stderr, "malloc() Failed");
		return -1;
	}

	x = (double*)malloc(size_x);
	if (x == NULL) {
		fprintf(stderr, "malloc() Failed");
		return -1;
	}

	y = (double*)malloc(size_y);
	if (y == NULL) {
		fprintf(stderr, "malloc() Failed");
		return -1;
	}

	init_array(x, y , A);

	printf("A:\n");

	for (i = 0; i < HEIGHT; i++) {										
		for (j = 0; j < WIDTH; j++) {
			printf("%f ", A[IDX2C(i, j, HEIGHT)]);
		}
		printf("\n");
	}

	printf(" Now normally \n");

	for (i = 0; i < HEIGHT; i++) {
		for (j = 0; j < WIDTH; j++) {
			printf("%f ", A[i*WIDTH + j] );
		}
		printf("\n");
	}

	printf("And vectors x , y\n");

	for (i = 0; i < WIDTH; i++) {
		printf("%f ", x[i]);
	}

	printf("\n");

	for (i = 0; i < HEIGHT; i++) {
		printf("%f ", y[i]);
	}

	printf("\n");

	/*-------------"Kernel" Preparation ------------*/

	double* d_A;
	double* d_x;
	double* d_y;

	cuda_status = hipMalloc((void **)&d_A, sizeA);
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "hipMalloc Failed!");
		goto Error;
	}

	cuda_status = hipMalloc((void **)&d_x, size_x);
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "hipMalloc Failed!");
		goto Error;
	}

	cuda_status = hipMalloc((void **)&d_y, size_y);
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "hipMalloc Failed!");
		goto Error;
	}

	/*----------- cuBLAS parameter setup ----------*/

	cuBl_stat = hipblasCreate(&handle);
	cuBl_stat = hipblasSetMatrix(HEIGHT, WIDTH, sizeof(*A), A, HEIGHT, d_A, HEIGHT);          //cp a->d_a
	cuBl_stat = hipblasSetVector(WIDTH, sizeof(*x), x, 1, d_x, 1);							 //cp x->d_x
	cuBl_stat = hipblasSetVector(HEIGHT, sizeof(*y), y, 1, d_y, 1);							 //cp y->d_y

	double alpha = 1.0;
	double beta = 0.0;

	// matrix -vector  multiplication:    d_y = alpha * d_a * d_x + beta * d_y
	// d_A - HEIGHT x WIDTH   matrix; d_x - WIDTH-vector , d_y - HEIGHT-vector;
	// alpha ,beta - scalars

	cuBl_stat = hipblasDgemv(handle, HIPBLAS_OP_N, HEIGHT, WIDTH, &alpha, d_A, HEIGHT, d_x, 1, &beta, d_y, 1);

	cuBl_stat = hipblasGetVector(HEIGHT, sizeof(*y), d_y, 1, y, 1);             //copy d_y -> y

	printf("y after  Dgemv ::\n");

	for (j = 0; j < HEIGHT; j++)
	{
		printf("%f ", y[j]);                                     //  print y after  dgemv
	}
	printf("\n");

	/*------ Now that we have calculated y = A * x we shall input that y in another Dgemv this time with A^T ---------*/

	cuBl_stat = hipblasDgemv(handle, HIPBLAS_OP_T, HEIGHT, WIDTH, &alpha, d_A, HEIGHT, d_y, 1, &beta, d_y, 1);

	cuBl_stat = hipblasGetVector(WIDTH, sizeof(*y), d_y, 1, y, 1);

	printf("y after second Dgemv ::\n");

	for (j = 0; j < WIDTH; j++)
	{
		printf("%f ", y[j]);                                     //  print y after  dgemv
	}
	printf("\n");

Error:
	hipFree(d_A);
	hipFree(d_x);
	hipFree(d_y);
	hipblasDestroy(handle);                                     //  destroy  CUBLAS  context
	free(A);                                                    // free  host  memory
	free(x);                                                    // free  host  memory
	free(y);                                                    // free  host  memory

	return 0;
}
	